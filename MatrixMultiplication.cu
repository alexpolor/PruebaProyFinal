#include "hip/hip_runtime.h"
// ==========================================================
//
// File: MatrixMultiplication.cu
// Author: Samantha Casillas
// Description: This file implements the multiplication of a // matrix by another Matrix of the same dimension using
// CUDA.
// The time this implementation takes will be used to calculate // the improvement obtained when compared to C on the server
// Copyright (c) 2021 by Tecnologico de Monterrey.
// All Rights Reserved. May be reproduced for any
// non-commercial purpose.
//
//avg time = 0.003 ms
// 2 300*300 matrixes
// =====================================================

#include <stdio.h>
#include <stdlib.h>
#include <limits.h>
#include <hip/hip_runtime.h>
#include "utils.h"

#define RENS    300//00
#define COLS    300//00
#define THREADS 256
#define BLOCKS	MMIN(32, (((RENS * COLS) / THREADS) + 1))

__global__ void matrix_vector(int *matrix1, int *matrix2, int *result) {
	int tid = threadIdx.x + (blockIdx.x * blockDim.x);
  int j,k, acum = 0;

  while (tid < RENS){
		for (j = 0; j < COLS; j++) {
			acum=0;
			for (k=0;k<RENS;k++){
				acum+=matrix1[(tid*COLS)+k]*matrix2[(k*RENS) +j];
			}
	    result[(tid*COLS)+j]=acum;
		}
		tid += blockDim.x * gridDim.x;
  }


}

int main(int argc, char* argv[]) {
	int i, j, *matrix1, *matrix2, *result;
  int *d_m1, *d_m2, *d_r;
	double ms;

	matrix1 = (int*) malloc(sizeof(int) * RENS * COLS);
	matrix2 = (int*) malloc(sizeof(int) * RENS * COLS);
	result = (int*) malloc(sizeof(int) * RENS * COLS);

  for (i = 0; i < RENS; i++) {
		for (j = 0; j < COLS; j++) {
			matrix1[(i * COLS) + j] = (j + 1);
			matrix2[(i * COLS) + j] = (j + 2);
		}
	}

  hipMalloc((void**)&d_m1, sizeof(int) * RENS* COLS);
  hipMalloc((void**)&d_m2, sizeof(int) * RENS* COLS);
  hipMalloc((void**)&d_r, sizeof(int) * RENS* COLS);

  hipMemcpy(d_m1, matrix1, sizeof(int) * RENS* COLS, hipMemcpyHostToDevice);
  hipMemcpy(d_m2, matrix2, sizeof(int) * RENS* COLS, hipMemcpyHostToDevice);

	printf("Starting...\n");
	ms = 0;
	for (i = 0; i < N; i++) {
		start_timer();

		matrix_vector<<<BLOCKS, THREADS>>>(d_m1, d_m2, d_r);

		ms += stop_timer();
	}

  hipMemcpy(result, d_r, sizeof(int) * RENS*COLS, hipMemcpyDeviceToHost);
/* //print the results matrix
	for (i = 0; i < RENS; i++) {
		for (j = 0; j < COLS; j++) {
			printf("%d, ", result[(i*COLS)+j]);
		}
		printf("\n" );
	}*/
	printf("avg time = %.5lf ms\n", (ms / N));

  hipFree(d_m1); hipFree(d_m2); hipFree(d_r);
	free(matrix1); free(matrix2); free(result);
	return 0;
}
